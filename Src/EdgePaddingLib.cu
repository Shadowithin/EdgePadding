#include "hip/hip_runtime.h"
﻿#include "EdgePaddingLib.h"

namespace EdgePadding {

    __device__ bool isZeroPixel(uchar4 p) {
        return (p.x == 0 && p.y == 0 && p.z == 0 && p.w == 0);
    }

    __global__ void fillZeroPixelsKernel(const uchar4* input, const uint8_t* input_mask, uchar4* output, uint8_t* output_mask, int width, int height, int* stillHasZero) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) return;

        int idx = y * width + x;
        uchar4 p = input[idx];
        uint8_t m = input_mask[idx];

        if (m > 0) {
            output[idx] = p;
            output_mask[idx] = m;
            return;
        }

        int sumX = 0, sumY = 0, sumZ = 0, sumW = 0;
        int count = 0;

        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                if (dx == 0 && dy == 0) continue;
                if (dx != 0 && dy != 0) continue;
                int nx = x + dx;
                int ny = y + dy;
                if (nx < 0 || ny < 0 || nx >= width || ny >= height) continue;

                uchar4 neighbor = input[ny * width + nx];
                uint8_t neighbor_mask = input_mask[ny * width + nx];
                if (neighbor_mask > 0) {
                    sumX += neighbor.x;
                    sumY += neighbor.y;
                    sumZ += neighbor.z;
                    sumW += neighbor.w;
                    count++;
                }
            }
        }

        if (count > 0) {
            output[idx].x = sumX / count;
            output[idx].y = sumY / count;
            output[idx].z = sumZ / count;
            output[idx].w = sumW / count;
            output_mask[idx] = 255;
        }
        else {
            output[idx] = p;
            output_mask[idx] = m;
            atomicAdd(stillHasZero, 1);
        }
    }

    __host__ int fillZeroPixels(const uchar4* input, const uint8_t* input_mask, uchar4* output, int width, int height) {

        int zeroCount = INT_MAX;
        int iter = 0;
        int maxIter = width + height;

        size_t imageSize = width * height * sizeof(uchar4);
        size_t maskSize = width * height * sizeof(uint8_t);

        uchar4* devImgA;
        uchar4* devImgB;
        uint8_t* devMaskA;
        uint8_t* devMaskB;
        int* devZeroCount;

        hipMalloc(&devImgA, imageSize);
        hipMalloc(&devImgB, imageSize);
        hipMalloc(&devMaskA, maskSize);
        hipMalloc(&devMaskB, maskSize);
        hipMalloc(&devZeroCount, sizeof(int));

        // 上传图像到 CUDA
        hipMemcpy(devImgA, input, imageSize, hipMemcpyHostToDevice);
        hipMemcpy(devMaskA, input_mask, maskSize, hipMemcpyHostToDevice);

        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        auto start = std::chrono::system_clock::now();
        for (int i = 0; i < maxIter && zeroCount > 0; i++)
        {
            zeroCount = 0;

            hipMemcpy(devZeroCount, &zeroCount, sizeof(int), hipMemcpyHostToDevice);

            EdgePadding::fillZeroPixelsKernel << <grid, block >> > (devImgA, devMaskA, devImgB, devMaskB, width, height, devZeroCount);
            hipDeviceSynchronize();

            hipMemcpy(&zeroCount, devZeroCount, sizeof(int), hipMemcpyDeviceToHost);
            //std::cout << "Iteration " << ++iter << ": remaining zero pixels = " << zeroCount << std::endl;

            std::swap(devImgA, devImgB);
            std::swap(devMaskA, devMaskB);

            iter = i + 1;
        }

        auto end = std::chrono::system_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << iter << " , " << duration.count() << std::endl;

        // 拷回主机查看结果（可选）
        hipMemcpy(output, devImgA, imageSize, hipMemcpyDeviceToHost);

        hipFree(devImgA);
        hipFree(devImgB);
        hipFree(devMaskA);
        hipFree(devMaskB);
        hipFree(devZeroCount);

        return 0;
    }
}

